#include "hip/hip_runtime.h"
#include "generate_op_cuda.cuh"


__global__ void img_op(double* z, unsigned char* data, double far_, double near_, float mask_threshold) {
    int global_index = threadIdx.x + blockDim.x * blockIdx.x;
    int total_index = 3 * global_index + 2;

    if (global_index < (HEIGHT* WIDTH)) {
        int z_jump = blockIdx.z * HEIGHT * WIDTH;
        int data_jump = HEIGHT * WIDTH * CHANNEL * blockIdx.z;

        unsigned char* mask_img = data + data_jump * 2;
        unsigned char* depth_img = data + data_jump;

        *(mask_img + total_index) = *(mask_img + total_index) > mask_threshold;

        *(z + global_index + z_jump) = 1. - (far_ - near_) * (((double)(*(depth_img + total_index)) / 255.) * (*(mask_img + total_index))) + near_;
#if DEBUG
        if (global_index == 250000) {
            printf("global idx : %d, z : %lf, mask_img : %d , depth_img : %d\n", \
                global_index, *(z + global_index), *(mask_img + total_index), *(depth_img + total_index));
        }
#endif // DEBUG
    }
}

__global__ void point_op(double* dst_points, unsigned char* dst_point_colors, unsigned char* src_rgb, double* src_z, double* src_inverse_k) {
    int blockIndex = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (blockIndex < WIDTH) {
        int global_index = blockIndex + blockIdx.y * WIDTH;
        int total_index = 3 * global_index;
        int data_jump = HEIGHT * WIDTH * CHANNEL * blockIdx.z;
        int z_jump = blockIdx.z * HEIGHT * WIDTH;
        double u = blockIndex;
        double v = blockIdx.y;

        if (*(src_z + global_index + z_jump) < 0) {
            *(dst_points + total_index + data_jump) = *(src_z + global_index + z_jump) * *(src_inverse_k) * u + \
                *(src_z + global_index + z_jump) * *(src_inverse_k + 1) * v + \
                *(src_z + global_index + z_jump) * *(src_inverse_k + 2);
            *(dst_points + total_index + 1 + data_jump) = *(src_z + global_index + z_jump) * *(src_inverse_k + 3) * u + \
                *(src_z + global_index + z_jump) * *(src_inverse_k + 4) * v + \
                *(src_z + global_index + z_jump) * *(src_inverse_k + 5);
            *(dst_points + total_index + 2 + data_jump) = *(src_z + global_index + z_jump);

            *(dst_point_colors + total_index + data_jump) = *(src_rgb + total_index + 2 + data_jump);
            *(dst_point_colors + total_index + 1 + data_jump) = *(src_rgb + total_index + 1 + data_jump);
            *(dst_point_colors + total_index + 2 + data_jump) = *(src_rgb + total_index + data_jump);
        }
        else {
            *(dst_points + total_index + data_jump) = NULL;
            *(dst_points + total_index + 1 + data_jump) = NULL;
            *(dst_points + total_index + 2 + data_jump) = NULL;

            *(dst_point_colors + total_index + data_jump) = NULL;
            *(dst_point_colors + total_index + 1 + data_jump) = NULL;
            *(dst_point_colors + total_index + 2 + data_jump) = NULL;


        }
#if DEBUG
        if (blockIndex == 0) {
            printf("global idx : %d, z : %lf, point : %lf , rgb : %d\n", \
                global_index, *(src_z + global_index), *(dst_points + total_index),\
                *(dst_point_colors + total_index));

        }
#endif // DEBUG
    }
}
