#include "hip/hip_runtime.h"
#include "img_op.cuh"





// Helper function for using CUDA to add vectors in parallel.
hipError_t img_num_compare_call(bool* dst, unsigned char* src, int size, int compare_num) {
    unsigned char* dev_src = 0;
    bool* dev_dst = 0;
    
    hipError_t cudaStatus;

    int img_grid1D = ceil((float)size / (float)MAX_BLOCK_NUM);

    dim3 grid(img_grid1D, 1, 1);
    dim3 block(MAX_BLOCK_NUM, 1, 1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_dst, size * sizeof(bool));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&dev_src, size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_src, src, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    cud::compare_bool<< <grid, block >> > (dev_dst, dev_src, compare_num);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(dst, dev_dst, size * sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_src);
    hipFree(dev_dst);

    return cudaStatus;
}

hipError_t img_automation_cuda(double** dst, unsigned char** src) {
    int pixel_size = HEIGHT * WIDTH * CHANNEL;

    unsigned char* src_buffer = (unsigned char*)malloc(pixel_size * sizeof(unsigned char));
    bool* dst_buffer = (bool*)malloc(pixel_size * sizeof(bool));



    free(src_buffer);
    free(dst_buffer);
}
