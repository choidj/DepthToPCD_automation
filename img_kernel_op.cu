#include "hip/hip_runtime.h"
#include "img_op_cuda.cuh"


__global__ void compare_bool(bool* dst, const unsigned char* src, const int compare_num) {
    int blockIndex = threadIdx.x;
    int globalIndex = threadIdx.x + blockDim.x * blockIdx.x;

    *(dst + globalIndex) = *(src + globalIndex) > compare_num;
#if DEBUG
    if (blockIndex == 0) {
        printf("block idx : %d, src : %d,result : %d \n", blockIndex, *(src + globalIndex), *(dst + globalIndex));
    }
#endif // DEBUG


}