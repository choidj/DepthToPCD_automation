#include "hip/hip_runtime.h"
#include "transform_op_cuda.cuh"


__global__ void img_op(double* z, unsigned char* depth_img, unsigned char* mask_img, double far, double near, float mask_threshold) {
    int globalIndex = threadIdx.x + blockDim.x * blockIdx.x;


    *(mask_img + globalIndex) = *(mask_img + globalIndex) > mask_threshold;

    *(z + globalIndex) = 1. - (far - near)*(((double)(*(depth_img + globalIndex)) / 255.) * (*(mask_img + globalIndex))) + near;

#if DEBUG
    if (globalIndex == 250000) {
        printf("global idx : %d, z : %lf, mask_img : %d , depth_img : %d\n", globalIndex, *(z + globalIndex), *(mask_img + globalIndex), *(depth_img + globalIndex));
    }
#endif // DEBUG
}

__global__ void point_op(double* dst_points, unsigned char* dst_point_colors, unsigned char* src_rgb, double* src_z, double* src_inverse_k) {
    int blockIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (blockIndex < WIDTH) {
        int globalIndex = blockIndex + blockIdx.y * WIDTH;
        int u = blockIndex;
        int v = blockIdx.y;

        *(dst_points + (globalIndex * 3)) = *(src_z + globalIndex) * *(src_inverse_k) * u +  \
            *(src_z + globalIndex) * *(src_inverse_k + 1) * v + \
            *(src_z + globalIndex) * *(src_inverse_k + 2);
        *(dst_points + (globalIndex * 3) + 1) = *(src_z + globalIndex) * *(src_inverse_k + 4)*u + \
            * (src_z + globalIndex) * *(src_inverse_k + 5) * v + \
            * (src_z + globalIndex) * *(src_inverse_k + 6);
        *(dst_points + (globalIndex * 3) + 2) = *(src_z + globalIndex);

        *(dst_point_colors + (globalIndex * 3)) = *(src_rgb + globalIndex);
        *(dst_point_colors + (globalIndex * 3) + 1) = *(src_rgb + globalIndex + 1);
        *(dst_point_colors + (globalIndex * 3) + 2) = *(src_rgb + globalIndex + 2);


#if DEBUG
        if (globalIndex == 250000) {
            printf("global idx : %d, z : %lf, mask_img : %d , depth_img : %d\n", globalIndex, *(z + globalIndex), *(mask_img + globalIndex), *(depth_img + globalIndex));
        }

#endif // DEBUG
    }
}
