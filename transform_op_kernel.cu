#include "hip/hip_runtime.h"
#include "transform_op_cuda.cuh"


__global__ void img_op(double* z, unsigned char* depth_img, unsigned char* mask_img, double far_, double near_, float mask_threshold) {
    int global_index = threadIdx.x + blockDim.x * blockIdx.x;
    int total_index = 3 * global_index + 2;

    if (global_index < (HEIGHT* WIDTH)) {
        *(mask_img + total_index) = *(mask_img + total_index) > mask_threshold;

        *(z + global_index) = 1. - (far_ - near_) * (((double)(*(depth_img + total_index)) / 255.) * (*(mask_img + total_index))) + near_;
#if DEBUG
        if (global_index == 250000) {
            printf("global idx : %d, z : %lf, mask_img : %d , depth_img : %d\n", \
                global_index, *(z + global_index), *(mask_img + total_index), *(depth_img + total_index));
        }
#endif // DEBUG
    }
}

__global__ void point_op(double* dst_points, unsigned char* dst_point_colors, unsigned char* src_rgb, double* src_z, double* src_inverse_k) {
    int blockIndex = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (blockIndex < WIDTH) {
        int global_index = blockIndex + blockIdx.y * WIDTH;
        int total_index = 3 * global_index;

        double u = blockIndex;
        double v = blockIdx.y;

        if (*(src_z + global_index) < 0) {
            *(dst_points + total_index) = *(src_z + global_index) * *(src_inverse_k) * u + \
                *(src_z + global_index) * *(src_inverse_k + 1) * v + \
                *(src_z + global_index) * *(src_inverse_k + 2);
            *(dst_points + total_index + 1) = *(src_z + global_index) * *(src_inverse_k + 3) * u + \
                *(src_z + global_index) * *(src_inverse_k + 4) * v + \
                *(src_z + global_index) * *(src_inverse_k + 5);
            *(dst_points + total_index + 2) = *(src_z + global_index);

            *(dst_point_colors + total_index) = *(src_rgb + total_index + 2);
            *(dst_point_colors + total_index + 1) = *(src_rgb + total_index + 1);
            *(dst_point_colors + total_index + 2) = *(src_rgb + total_index);
        }
        else {
            *(dst_points + total_index) = NULL;
            *(dst_points + total_index + 1) = NULL;
            *(dst_points + total_index + 2) = NULL;

            *(dst_point_colors + total_index) = NULL;
            *(dst_point_colors + total_index + 1) = NULL;
            *(dst_point_colors + total_index + 2) = NULL;


        }
#if DEBUG
        if (blockIndex == 0) {
            printf("global idx : %d, z : %lf, point : %lf , rgb : %d\n", \
                global_index, *(src_z + global_index), *(dst_points + total_index),\
                *(dst_point_colors + total_index));

        }
#endif // DEBUG
    }
}
