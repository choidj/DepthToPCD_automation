#include "hip/hip_runtime.h"
#include "transform_op_cuda.cuh"


__global__ void img_op(double* z, unsigned char* depth_img, unsigned char* mask_img, double far_, double near_, float mask_threshold) {
    int globalIndex = threadIdx.x + blockDim.x * blockIdx.x;


    *(mask_img + globalIndex) = *(mask_img + globalIndex) > mask_threshold;

    *(z + globalIndex) = 1. - (far_ - near_)*(((double)(*(depth_img + globalIndex)) / 255.) * (*(mask_img + globalIndex))) + near_;

#if DEBUG
    if (globalIndex == 250000) {
        printf("global idx : %d, z : %lf, mask_img : %d , depth_img : %d\n", globalIndex, *(z + globalIndex), *(mask_img + globalIndex), *(depth_img + globalIndex));
    }
#endif // DEBUG
}

__global__ void point_op(double* dst_points, unsigned char* dst_point_colors, unsigned char* src_rgb, double* src_z, double* src_inverse_k) {
    int blockIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (blockIndex < WIDTH) {
        int globalIndex = blockIndex + blockIdx.y * WIDTH;
        int u = blockIndex;
        int v = blockIdx.y;
        if (*(src_z + globalIndex) > 0) {
            *(dst_points + globalIndex) = *(src_z + globalIndex) * *(src_inverse_k)*u + \
                *(src_z + globalIndex) * *(src_inverse_k + 1) * v + \
                *(src_z + globalIndex) * *(src_inverse_k + 2);
            *(dst_points + globalIndex + HEIGHT * WIDTH) = *(src_z + globalIndex) * *(src_inverse_k + 4) * u + \
                *(src_z + globalIndex) * *(src_inverse_k + 5) * v + \
                *(src_z + globalIndex) * *(src_inverse_k + 6);
            *(dst_points + globalIndex + HEIGHT * WIDTH * 2) = *(src_z + globalIndex);

            *(dst_point_colors + globalIndex) = *(src_rgb + globalIndex);
            *(dst_point_colors + globalIndex + HEIGHT * WIDTH) = *(src_rgb + globalIndex + 1);
            *(dst_point_colors + globalIndex + HEIGHT * WIDTH * 2) = *(src_rgb + globalIndex + 2);
        }
        else {
            *(dst_points + globalIndex) = -100;
            *(dst_points + globalIndex + HEIGHT * WIDTH) = -100;
            *(dst_points + globalIndex + HEIGHT * WIDTH * 2) = -100;

            *(dst_point_colors + globalIndex) = 0;
            *(dst_point_colors + globalIndex + HEIGHT * WIDTH) = 0;
            *(dst_point_colors + globalIndex + HEIGHT * WIDTH * 2) = 0;
        }
#if DEBUG
        if (globalIndex == 250000) {
            printf("global idx : %d, z : %lf, mask_img : %d , depth_img : %d\n", globalIndex, *(src_z + globalIndex), *(dst_points + globalIndex), *(dst_point_colors + globalIndex));
        }
#endif // DEBUG
    }
}