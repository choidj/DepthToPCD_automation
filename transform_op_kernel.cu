#include "hip/hip_runtime.h"
#include "transform_op_cuda.cuh"


__global__ void img_op(double* z, unsigned char* depth_img, unsigned char* mask_img, double far, double near, float mask_threshold) {
    int globalIndex = threadIdx.x + blockDim.x * blockIdx.x;


    *(mask_img + globalIndex) = *(mask_img + globalIndex) > mask_threshold;

    *(z + globalIndex) = 1. - (far - near)*(((double)(*(depth_img + globalIndex)) / 255.) * (*(mask_img + globalIndex))) + near;

#if DEBUG
    if (globalIndex == 250000) {
        printf("global idx : %d, z : %lf, mask_img : %d , depth_img : %d\n", globalIndex, *(z + globalIndex), *(mask_img + globalIndex), *(depth_img + globalIndex));
    }
#endif // DEBUG
}