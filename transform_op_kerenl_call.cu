#include "hip/hip_runtime.h"
#include "transform_op_cuda.cuh"


// Helper function for using CUDA to add vectors in parallel.
hipError_t img_op_kernel_call(double* dst_z, unsigned char* src_depth_img, unsigned char* src_mask_img) {
    double* dev_z; unsigned char* dev_depth_img; unsigned char* dev_mask_img;

    double far_ = 5; double near_ = 0.3; float mask_threshold = 10;

    int size = HEIGHT * WIDTH;

    hipError_t cudaStatus;

    int img_grid1D = ceil((float)size / (float)MAX_BLOCK_NUM);

    dim3 grid(img_grid1D, 1, 1); dim3 block(MAX_BLOCK_NUM, 1, 1);

#if DEBUG
    for (int i = 0; i < 30; i++) {
        printf("[i : %d] depth src : %d, mask src : %d\n", i, *(src_depth_img +i), *(src_mask_img + i));
    }
#endif

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_z, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_z!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&dev_depth_img, CHANNEL * size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_depth_img!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_mask_img, CHANNEL * size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_mask_img!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_depth_img, src_depth_img, CHANNEL * size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_depth_img!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_mask_img, src_mask_img, CHANNEL * size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed dev_mask_img!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    img_op << <grid, block >> > (dev_z, dev_depth_img, dev_mask_img, far_, near_, mask_threshold);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(dst_z, dev_z, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on z!");
        goto Error;
    }

Error:
    hipFree(dev_z); hipFree(dev_depth_img); hipFree(dev_mask_img);

    return cudaStatus;
}

hipError_t point_op_kernel_call(double** dst_points, unsigned char** dst_point_colors, unsigned char* src_rgb, double* src_z) {
    // need to add inverse op???
    double inverse_k[][3] = {
        {-0.00174699220352319, 0 ,0.559037505127422},
        {0, -0.00174346879155994, 0.418432509974385},
        {0, 0, 1}
    };
    int k_size = 9;

    double* dev_inverse_k; double* dev_z; double* dev_points; unsigned char* dev_point_colors; unsigned char* dev_rgb;

    int size = HEIGHT * WIDTH;

    hipError_t cudaStatus;

    int grid2D_x = ceil((float)WIDTH / (float)NORM_BLOCK_NUM);
    int grid2D_y = HEIGHT;

    dim3 grid(grid2D_x, grid2D_y, 1); dim3 block(NORM_BLOCK_NUM, 1, 1);

#if DEBUG
    int inx = 0;
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            if (*(src_z + i * WIDTH + j) != 1.3) {
                printf("height : %d, width : %d, rgb src : %d, z src : %lf\n", i, j, *(src_rgb + i), *(src_z + i * WIDTH + j));
                if (*(src_z + i * WIDTH + j) < 0)
                    inx++;
            }
        }
    }
    printf("total - point : %d\n", inx);
#endif

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_z, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_z!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rgb, CHANNEL * size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_rgb!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_points, CHANNEL * size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_points!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_point_colors, CHANNEL * size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_point_colors!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inverse_k, k_size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_inverse_k!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_inverse_k, inverse_k, k_size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_inverse_k!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_z, src_z, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed dev_z!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_rgb, src_rgb, CHANNEL * size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed dev_rgb!");
        goto Error;
    }
    
    // Launch a kernel on the GPU with one thread for each element.
    point_op << <grid, block >> > (dev_points, dev_point_colors, dev_rgb, dev_z, dev_inverse_k);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(*dst_points, dev_points, CHANNEL * size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on z!");
        goto Error;
    }

    cudaStatus = hipMemcpy(*dst_point_colors, dev_point_colors, CHANNEL * size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on z!");
        goto Error;
    }

Error:
    hipFree(dev_z); hipFree(dev_inverse_k); 
    hipFree(dev_points); hipFree(dev_point_colors);
    hipFree(dev_rgb);

    return cudaStatus;
}


// depth image : *(src), rgb image : *(src + 1), mask image : *(src + 2)
void trans_automation_cuda(double** dst_point, unsigned char** dst_point_color, unsigned char** src_images) {
    hipError_t cudaStatus;
    int pixel_size = HEIGHT * WIDTH * CHANNEL;
    double* z = (double*)malloc(HEIGHT * WIDTH * sizeof(double));

#if DEBUG
    for (int i = 0; i < 30; i++) {
        printf("[i : %d] depth src : %d, rgb src : %d, mask src : %d\n", i, *(*(src_images) + i), *(*(src_images + 1) + i), *(*(src_images + 2) + i));
    }
#endif

    // image operation kernel call  matlab :-- pts = zeros(height*width, 3) color = uint8(zeros(height * width, 3))--
    cudaStatus = img_op_kernel_call(z, *(src_images), *(src_images + 2));

    //point operation kernel call here..
    cudaStatus = point_op_kernel_call(dst_point, dst_point_color, *(src_images + 1), z);

    free(z);
}
