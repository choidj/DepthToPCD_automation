#include "hip/hip_runtime.h"
#include "transform_op_cuda.cuh"

// Helper function for using CUDA to add vectors in parallel.
hipError_t img_op_kernel_call(double* z, unsigned char* depth_img, unsigned char* mask_img) {
    double* dev_z; unsigned char* dev_depth_img; unsigned char* dev_mask_img;

    double far = 5; double near = 0.3; float mask_threshold = 10;

    int size = HEIGHT * WIDTH;
    hipError_t cudaStatus;

    int img_grid1D = ceil((float)size / (float)MAX_BLOCK_NUM);

    dim3 grid(img_grid1D, 1, 1); dim3 block(MAX_BLOCK_NUM, 1, 1);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_z, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_z!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&dev_depth_img, size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_depth_img!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_mask_img, size * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed on dev_mask_img!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_depth_img, depth_img, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on dev_depth_img!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_depth_img, depth_img, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed dev_mask_img!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    img_op << <grid, block >> > (dev_z, dev_depth_img, dev_mask_img, far, near, mask_threshold);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(z, dev_z, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed on z!");
        goto Error;
    }

Error:
    hipFree(dev_z); hipFree(dev_depth_img); hipFree(dev_mask_img);

    return cudaStatus;
}


// depth image : *(src), rgb image : *(src + 1), mask image : *(src + 2)
hipError_t trans_automation_cuda(double** dst, unsigned char** src) {
    hipError_t cudaStatus;
    int pixel_size = HEIGHT * WIDTH * CHANNEL;

    double inverse_k[][3] = { 
        {-0.00174699220352319, 0 ,0.559037505127422},
        {0, -0.00174346879155994, 0.418432509974385},
        {0, 0, 1}
    };


    double* z = (double*)malloc(HEIGHT * WIDTH * sizeof(double));



    // image operation kernel call  matlab :-- pts = zeros(height*width, 3) color = uint8(zeros(height * width, 3))--
    cudaStatus = img_op_kernel_call(z, *(src + 1), *(src + 2));

    //point operation kernel call



    std::free(z);
    for (int i = 0; i < 3; i++)
        delete(inverse_k[i]);
    delete(inverse_k);

    return cudaStatus;
}
